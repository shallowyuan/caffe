#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/vision_layers.hpp"

namespace caffe {

template <typename Dtype>
__global__ void slice_map_forward_gpu_kernel(
    const Dtype* bottom_data, const int height_b, const int width_b,
    const int y_begin, const int x_begin,
    const int channels, const int height, const int width,
    Dtype* top_data) {
  CUDA_KERNEL_LOOP(index, channels * height * width) {
    int x = index % width;
    int y_ind = index / width;
    int y = y_ind % height;
    int c = y_ind / height;
    top_data[index] = bottom_data[
        (c * height_b + (y + y_begin)) * width_b + x + x_begin];
  }
}

template <typename Dtype>
void slice_map_forward_gpu(
    const Dtype* bottom_data, const int height_b, const int width_b,
    const int y_begin, const int x_begin,
    const int channels, const int height, const int width,
    Dtype* top_data) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  slice_map_forward_gpu_kernel<Dtype><<<CAFFE_GET_BLOCKS(
    channels * height * width), CAFFE_CUDA_NUM_THREADS>>>(
    bottom_data, height_b, width_b,
    y_begin, x_begin,
    channels, height, width,
    top_data);
  CUDA_POST_KERNEL_CHECK;
}

template <typename Dtype>
__global__ void slice_map_backward_gpu_kernel(
    const Dtype* top_diff,
    const int channels, const int height, const int width,
    const int height_b, const int width_b,
    const int y_begin, const int x_begin,
    Dtype* bottom_diff) {
  CUDA_KERNEL_LOOP(index, channels * height * width) {
    int x = index % width;
    int y_ind = index / width;
    int y = y_ind % height;
    int c = y_ind / height;
    bottom_diff[(c * height_b + (y + y_begin)) * width_b + x + x_begin] = \
        top_diff[index];
  }
}

template <typename Dtype>
void slice_map_backward_gpu(
    const Dtype* top_diff,
    const int channels, const int height, const int width,
    const int height_b, const int width_b,
    const int y_begin, const int x_begin,
    Dtype* bottom_diff) {
  caffe_gpu_set(channels  * height_b * width_b, Dtype(0), bottom_diff);
  // NOLINT_NEXT_LINE(whitespace/operators)
  slice_map_backward_gpu_kernel<Dtype><<<CAFFE_GET_BLOCKS(
    channels * height * width), CAFFE_CUDA_NUM_THREADS>>>(
    top_diff,
    channels, height, width,
    height_b, width_b,
    y_begin, x_begin,
    bottom_diff);
  CUDA_POST_KERNEL_CHECK;
}


template <typename Dtype>
void SliceMapLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->mutable_gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  for (int n = 0; n < num_; n++) {
    slice_map_forward_gpu(
        bottom_data + bottom[0]->offset(n),
        bottom[0]->height(), bottom[0]->width(),
        y_begin_, x_begin_,
        channels_, height_, width_,
        top_data + top[0]->offset(n));
  }
}

template <typename Dtype>
void SliceMapLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  if (!propagate_down[0]) { return; }
  Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
  const Dtype* top_diff = top[0]->gpu_diff();
  for (int n = 0; n < num_; n++) {
    slice_map_backward_gpu(
        top_diff + top[0]->offset(n),
        channels_, height_, width_,
        bottom[0]->height(), bottom[0]->width(),
        y_begin_, x_begin_,
        bottom_diff + bottom[0]->offset(n));
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(SliceMapLayer);

}  // namespace caffe
