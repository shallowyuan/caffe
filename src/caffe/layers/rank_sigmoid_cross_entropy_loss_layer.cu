#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/vision_layers.hpp"

namespace caffe {

// sxe sigmoid cross entropy
template <typename Dtype>
__global__ void diff_rank_sxe_kernel(
    const int n, const Dtype*sigmoid, const Dtype* target, Dtype *bottom_diff) {
  CUDA_KERNEL_LOOP(index, n) {
    bottom_diff[index] = (target[index] >= 0) * (sigmoid[index] - 1) +
          + (target[index] <= 0) * sigmoid[index];
  }
}
template <typename Dtype>
void gpu_diff_rank_sxe(const int n, const Dtype*sigmoid, const Dtype* target,
  Dtype *bottom_diff) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  diff_rank_sxe_kernel<Dtype><<<CAFFE_GET_BLOCKS(n), CAFFE_CUDA_NUM_THREADS>>>(
      n, sigmoid, target, bottom_diff);
}

template <typename Dtype>
void RankSigmoidCrossEntropyLossLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
  // The forward pass computes the sigmoid outputs.
  sigmoid_bottom_vec_[0] = bottom[0];
  sigmoid_layer_->Forward(sigmoid_bottom_vec_, sigmoid_top_vec_);
  // Compute the loss (negative log likelihood)
  const int count = bottom[0]->count();
  const int num = bottom[0]->num();
  // Stable version of loss computation from input data
  const Dtype* input_data = bottom[0]->cpu_data();
  const Dtype* target = bottom[1]->cpu_data();
  Dtype loss = 0;
  for (int i = 0; i < count; ++i) {
    const Dtype loss_const =
        log(1 + exp(input_data[i] - 2 * input_data[i] * (input_data[i] >= 0)));
    const Dtype ploss = input_data[i] * (1 - (input_data[i] >= 0)) - loss_const;
    const Dtype nloss = input_data[i] * (- (input_data[i] >= 0)) - loss_const;
    // if t=0, both losses are added.
    loss -= (target[i] >= 0) * ploss + (target[i] <= 0) * nloss;
  }
  top[0]->mutable_cpu_data()[0] = loss / num;
}

template <typename Dtype>
void RankSigmoidCrossEntropyLossLayer<Dtype>::Backward_gpu(
    const vector<Blob<Dtype>*>& top, const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  if (propagate_down[1]) {
    LOG(FATAL) << this->type()
               << " Layer cannot backpropagate to label inputs.";
  }
  if (propagate_down[0]) {
    // First, compute the diff
    const int count = bottom[0]->count();
    const int num = bottom[0]->num();
    const Dtype* sigmoid_output_data = sigmoid_output_->gpu_data();
    const Dtype* target = bottom[1]->gpu_data();
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    gpu_diff_rank_sxe<Dtype>(count, sigmoid_output_data, target, bottom_diff);
    // Scale down gradient
    const Dtype loss_weight = top[0]->cpu_diff()[0];
    caffe_gpu_scal(count, loss_weight / num, bottom_diff);
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(RankSigmoidCrossEntropyLossLayer);


}  // namespace caffe
